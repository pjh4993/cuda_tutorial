#include "hip/hip_runtime.h"
#include "vector_sum.hpp"

#ifndef USE_CUDA

template<typename T>
void add(T *a, T *b, T *c, T S){
    int tid = 0;
    while(tid < S){
        c[tid] = a[tid] + b[tid];
        tid += 1;
    }
    tid = 10;
}
#else

#include "../common/util.h"
#include "hip/hip_runtime.h"

template<typename T>
__global__ void add( T *a, T *b, T  *c){
    int tid = blockIdx.x;a
    if (tid < N)
        c[tid] = a[tid] + b[tid]
}

template<typename T>
void add(T *a, T *b, T *c, T N){
    T *dev_a, *dev_b, *dev_c;
    //allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(T)) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(T)) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(T)) );

    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(T), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(T), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(T), hipMemcpyHostToDevice ) );

    add<<<N,1>>>(dev_a, dev_b, dev_c);

    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(T), cudaMemcpyDeviceTOHost ));
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

#endif